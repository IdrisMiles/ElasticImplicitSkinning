#include "../cuda_inc/cudatexture.h"

//CudaTexture::CudaTexture()
//{

//}


//CudaTexture::~CudaTexture()
//{
//    hipDestroyTextureObject(d_cuTex);
//    hipFreeArray(d_cuArray);
//}


//template<typename T>
//void CudaTexture::CreateCudaTexture(unsigned int _dim, T *_data)
//{
//    // Initialise cuda array
//    hipChannelFormatDesc channelDesc = hipCreateChannelDesc<T>();// hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
//    hipMalloc3DArray(&d_cuArray, &channelDesc, make_hipExtent(_dim*sizeof(T), _dim, _dim));


//    // Upload data to device here
//    hipMemcpy3DParms copy3DParams = {0};
//    copy3DParams.srcPtr = make_hipPitchedPtr((void*)_data, _dim*sizeof(T), _dim, _dim);
//    copy3DParams.dstArray = d_cuArray;
//    copy3DParams.extent = make_hipExtent(_dim, _dim, _dim);
//    copy3DParams.kind = hipMemcpyHostToDevice;
//    hipMemcpy3D(&copy3DParams);


//    // Initalise cuda texture
//    struct hipResourceDesc resDesc;
//    memset(&resDesc, 0, sizeof(resDesc));
//    resDesc.resType = hipResourceTypeArray;
//    resDesc.res.array.array = d_cuArray;

//    struct hipTextureDesc texDesc;
//    memset(&texDesc, 0, sizeof(texDesc));
//    texDesc.addressMode[0] = hipAddressModeClamp;
//    texDesc.addressMode[1] = hipAddressModeClamp;
//    texDesc.addressMode[2] = hipAddressModeClamp;
//    texDesc.filterMode = hipFilterModeLinear;
//    texDesc.readMode = hipReadModeElementType;
//    texDesc.normalizedCoords = 1;

//    d_cuTex = 0;
//    hipCreateTextureObject(&d_cuTex, &resDesc, &texDesc, NULL);
//}
