#include "hip/hip_runtime.h"
#include "ImplicitSkinKernels.h"
#include <hip/hip_runtime.h>
#include <stdio.h>


//------------------------------------------------------------------------------------------------
// CUDA Device Functions
//------------------------------------------------------------------------------------------------

//------------------------------------------------------------------------------------------------

__device__ glm::vec3 ProjectPointOnToPlane(const glm::vec3 &_point, const glm::vec3 &_planeOrigin, const glm::vec3 &_planeNormal)
{
    return (_point - (glm::dot(_point - _planeOrigin, _planeNormal) * _planeNormal));
}

//------------------------------------------------------------------------------------------------

__device__ void EvalGlobalField(float &_outputF,
                                const glm::vec3 &_samplePoint,
                                const int _numSamples,
                                const glm::mat4 *_textureSpace,
                                const glm::mat4 *_rigidTransforms,
                                const hipTextureObject_t *_fieldFuncs,
                                const int _numFields,
                                const hipTextureObject_t *_compOps,
                                const hipTextureObject_t *_theta,
                                const int _numOps,
                                const ComposedFieldCuda *_compFields,
                                const int _numCompFields)
{
    float f[100];
    glm::vec3 df[100];
    int i=0;
    for(i=0; i<_numFields; i++)
    {
        glm::mat4 rigidTrans = _rigidTransforms[i];
        glm::mat4 textureSpace = _textureSpace[i];
        glm::vec3 transformedPoint = glm::vec3(glm::inverse(rigidTrans) * glm::vec4(_samplePoint, 1.0f));
        glm::vec3 texturePoint = glm::vec3(textureSpace * glm::vec4(transformedPoint, 1.0f));
        texturePoint = 1.015f*texturePoint;

        float4 val = tex3D<float4>(_fieldFuncs[i], texturePoint.x, texturePoint.y, texturePoint.z);
        f[i] = val.w;
        df[i] = glm::vec3(val.x, val.y, val.z);
    }


    float cf[100];
    float maxF = FLT_MIN;
    for(i=0; i<_numCompFields; i++)
    {
        int f1Id = _compFields[i].fieldFuncA;
        int f2Id = _compFields[i].fieldFuncB;
        int coId = _compFields[i].compOp;

        if(f2Id > -1)
        {
            glm::vec3 df1(df[f1Id].x, df[f1Id].y, df[f1Id].z);
            glm::vec3 df2(df[f2Id].x, df[f2Id].y, df[f2Id].z);
            float angle = glm::angle(df1, df2);
            float theta = tex1D<float>(_theta[coId], (angle*0.5f*M_1_PI));

            float4 val = tex3D<float4>(_compOps[coId], f[f1Id], f[f2Id], theta);
            cf[i] = val.w;
        }
        else
        {
            cf[i] = f[f1Id];
        }
        maxF = (cf[i]>maxF) ? cf[i] : maxF;
    }


    _outputF = maxF;

}

//------------------------------------------------------------------------------------------------


__device__ void EvalGradGlobalField(float &_outputF,
                                    glm::vec3 &_outputG,
                                    const glm::vec3 &_samplePoint,
                                    const int _numSamples,
                                    const glm::mat4 *_textureSpace,
                                    const glm::mat4 *_rigidTransforms,
                                    const hipTextureObject_t *_fieldFuncs,
                                    const int _numFields,
                                    const hipTextureObject_t *_compOps,
                                    const hipTextureObject_t *_theta,
                                    const int _numOps,
                                    const ComposedFieldCuda *_compFields,
                                    const int _numCompFields)
{

    float h = 60.0f / 64.0f;

    float newf;
    EvalGlobalField(newf, _samplePoint, _numSamples, _textureSpace, _rigidTransforms, _fieldFuncs, _numFields, _compOps, _theta, _numOps, _compFields, _numCompFields);
    float x2;
    glm::vec3 sampleX = _samplePoint + glm::vec3(h, 0.0f, 0.0f);
    EvalGlobalField(x2, sampleX, _numSamples, _textureSpace, _rigidTransforms, _fieldFuncs, _numFields, _compOps, _theta, _numOps, _compFields, _numCompFields);
    float y2;
    glm::vec3 sampleY = _samplePoint + glm::vec3(0.0f, h, 0.0f);
    EvalGlobalField(y2, sampleY, _numSamples, _textureSpace, _rigidTransforms, _fieldFuncs, _numFields, _compOps, _theta, _numOps, _compFields, _numCompFields);
    float z2;
    glm::vec3 sampleZ = _samplePoint + glm::vec3(0.0f, 0.0f, h);
    EvalGlobalField(z2, sampleZ, _numSamples, _textureSpace, _rigidTransforms, _fieldFuncs, _numFields, _compOps, _theta, _numOps, _compFields, _numCompFields);

    _outputF = newf;
    _outputG = glm::vec3((x2-newf)/h, (y2-newf)/h, (z2-newf)/h);

}



//------------------------------------------------------------------------------------------------

__device__ void VertexProjection(glm::vec3 &_deformedVert,
                                 const float &_origIso,
                                 const float &_newIso,
                                 const glm::vec3 &_newIsoGrad,
                                 glm::vec3 &_prevIsoGrad,
                                 float &_gradAngle,
                                 const float &_sigma,
                                 const float &_contactAngle)
{
    float angle = _gradAngle = glm::degrees(glm::angle(glm::normalize(_newIsoGrad), glm::normalize(_prevIsoGrad)));

    if(angle <= _contactAngle)
    {
        glm::vec3 displacement = ( _sigma * (_newIso - _origIso) * (_newIsoGrad / glm::length2(_newIsoGrad)));
        _deformedVert = _deformedVert + displacement;
    }
}

//------------------------------------------------------------------------------------------------

__device__ void TangentialRelaxation (glm::vec3 &_deformedVert,
                                     const glm::vec3 &_normal,
                                     const float _origIso,
                                     const float _newIso,
                                      glm::vec3 *_verts,
                                     const int *_oneRingNeigh,
                                     const float *_centroidWeights,
                                     const int _numNeighs)
{
    float mu = 1.0f - powf(fabs(_newIso- _origIso) - 1.0f, 4.0f);
    mu = max(mu, 0.0f);

    glm::vec3 norm(0.0f, 0.0f, 0.0f);
    for(int i=0; i<_numNeighs; i++)
    {
        int nextNeigh = ((i+1)%_numNeighs);
        glm::vec3 neighVert = _verts[_oneRingNeigh[i]];
        glm::vec3 nextNeighVert = _verts[_oneRingNeigh[nextNeigh]];
        glm::vec3 faceNorm = glm::cross(neighVert - _deformedVert, nextNeighVert-_deformedVert);
        norm += faceNorm;
    }
    norm = glm::normalize(norm);

    glm::vec3 sumWeightedCentroid(0.0f);
    for(int i=0; i<_numNeighs; i++)
    {
        glm::vec3 neighVert = _verts[_oneRingNeigh[i]];
        glm::vec3 projNeighVert = ProjectPointOnToPlane(neighVert, _deformedVert, norm);//_normal);
        float barycentricCoord = _centroidWeights[i];
        sumWeightedCentroid += barycentricCoord * projNeighVert;
    }

    _deformedVert = ((1.0f - mu) * _deformedVert) + (mu * sumWeightedCentroid);
}

//------------------------------------------------------------------------------------------------

__device__ void LaplacianSmoothing(glm::vec3 &_deformedVert,
                                   const glm::vec3 &_normal,
                                   const int *_oneRingNeigh,
                                   const float *_centroidWeights,
                                   const int _numNeighs,
                                   const glm::vec3 *_verts,
                                   const float _beta)
{
    glm::vec3 centroid(0.0f, 0.0f, 0.0f);

    for(int i=0; i<_numNeighs; i++)
    {
        centroid += (_centroidWeights[i] * _verts[_oneRingNeigh[i]]);
    }

    _deformedVert = ((1.0f - _beta) * _deformedVert) + (_beta * centroid);
}


//------------------------------------------------------------------------------------------------
// CUDA Global Kernels
//------------------------------------------------------------------------------------------------

__global__ void EvalGlobalField_Kernel(float *_output,
                                       const glm::vec3 *_samplePoint,
                                       const int _numSamples,
                                       const glm::mat4 *_textureSpace,
                                       const glm::mat4 *_rigidTransforms,
                                       const hipTextureObject_t *_fieldFuncs,
                                       const int _numFields,
                                       const hipTextureObject_t *_compOps,
                                       const hipTextureObject_t *_theta,
                                       const int _numOps,
                                       const ComposedFieldCuda *_compFields,
                                       const int _numCompFields)
{
    int tid = threadIdx.x + (blockIdx.x * blockDim.x);

    if(tid >= _numSamples)
    {
        return;
    }

    EvalGlobalField(_output[tid], _samplePoint[tid], _numSamples,
                    _textureSpace, _rigidTransforms, _fieldFuncs, _numFields,
                    _compOps, _theta, _numOps,
                    _compFields, _numCompFields);
}

//------------------------------------------------------------------------------------------------

__global__ void EvalGradGlobalField_Kernel(float *_output,
                                           glm::vec3 *_outputG,
                                       const glm::vec3 *_samplePoint,
                                       const int _numSamples,
                                       const glm::mat4 *_textureSpace,
                                       const glm::mat4 *_rigidTransforms,
                                       const hipTextureObject_t *_fieldFuncs,
                                       const int _numFields,
                                       const hipTextureObject_t *_compOps,
                                       const hipTextureObject_t *_theta,
                                       const int _numOps,
                                       const ComposedFieldCuda *_compFields,
                                       const int _numCompFields)
{
    int tid = threadIdx.x + (blockIdx.x * blockDim.x);

    if(tid < _numSamples)
    {
        float f=0.0f;
        glm::vec3 grad=glm::vec3(0.0f, 0.0f, 0.0f);
        glm::vec3 sample = _samplePoint[tid];

        EvalGradGlobalField(f, grad, sample, _numSamples,
                            _textureSpace, _rigidTransforms, _fieldFuncs, _numFields,
                            _compOps, _theta, _numOps,
                            _compFields, _numCompFields);


//        printf("%i\n",tid);



        _output[tid] = f;
        _outputG[tid] = grad;
    }
}


//------------------------------------------------------------------------------------------------

__global__ void LinearBlendWeightSkin_Kernel(glm::vec3 *_deformedVert,
                                             const glm::vec3 *_origVert,
                                             glm::vec3 *_deformedNorms,
                                             const glm::vec3 *_origNorms,
                                             const glm::mat4 *_transform,
                                             const uint *_boneId,
                                             const float *_weight,
                                             const int _numVerts,
                                             const int _numBones)
{
    int tid = threadIdx.x + (blockIdx.x * blockDim.x);

    if(tid >= _numVerts)
    {
        return;
    }

    glm::mat4 boneTransform = glm::mat4(0.0f);

    float totalWeight = 0.0f;
    for(int i=0; i<4; i++)
    {
        uint boneId = _boneId[(tid*4) + i];
        float w = _weight[(tid*4) + i];
        boneTransform += (_transform[boneId] * w);

        totalWeight+=w;
    }

    _deformedVert[tid] = glm::vec3(boneTransform * glm::vec4(_origVert[tid], 1.0f));
    _deformedNorms[tid] = glm::transpose(glm::inverse(glm::mat3(boneTransform))) * _origNorms[tid];
}

//------------------------------------------------------------------------------------------------

__global__ void VertexProjection_Kernel(glm::vec3 *_deformedVert,
                                        const glm::vec3 *_normal,
                                        const float *_origIsoValue,
                                        glm::vec3 *_prevIsoGrad,
                                        const int _numVerts,
                                        const glm::mat4 *_textureSpace,
                                        const glm::mat4 *_rigidTransforms,
                                        const hipTextureObject_t *_fieldFuncs,
                                        const int _numFields,
                                        const hipTextureObject_t *_compOps,
                                        const hipTextureObject_t *_theta,
                                        const int _numOps,
                                        const ComposedFieldCuda *_compFields,
                                        const int _numCompFields,
                                        const float _sigma,
                                        const float _contactAngle)
{

    int tid = threadIdx.x + (blockIdx.x * blockDim.x);

    if(tid >= _numVerts)
    {
        return;
    }

    //----------------------------------------------------
    // Get iso value from global field
    glm::vec3 deformedVert = _deformedVert[tid];
    float origIsoValue = _origIsoValue[tid];
    glm::vec3 prevGrad = _prevIsoGrad[tid];
    glm::vec3 newGrad = glm::vec3(0.0f, 0.0f, 0.0f);
    float newIsoValue = 0.0f;

    EvalGradGlobalField(newIsoValue, newGrad, deformedVert, _numVerts,
                        _textureSpace, _rigidTransforms, _fieldFuncs, _numFields,
                        _compOps, _theta, _numOps,
                        _compFields, _numCompFields);


    //----------------------------------------------------
    // Perform vertex projection along gradient of global field
    float gradAngle;

    VertexProjection(deformedVert, origIsoValue, newIsoValue, newGrad, prevGrad, gradAngle, _sigma, _contactAngle);
    prevGrad = newGrad;

    _deformedVert[tid] = deformedVert;
    _prevIsoGrad[tid] = newGrad;

}

//------------------------------------------------------------------------------------------------

__global__ void TangentialRelaxation_Kernel(glm::vec3 *_deformedVert,
                                            const glm::vec3 *_normal,
                                            const float *_origIsoValue,
                                            glm::vec3 *_prevIsoGrad,
                                            const int _numVerts,
                                            const glm::mat4 *_textureSpace,
                                            const glm::mat4 *_rigidTransforms,
                                            const hipTextureObject_t *_fieldFuncs,
                                            const int _numFields,
                                            const hipTextureObject_t *_compOps,
                                            const hipTextureObject_t *_theta,
                                            const int _numOps,
                                            const ComposedFieldCuda *_compFields,
                                            const int _numCompFields,
                                            const int *_oneRingVerts,
                                            const float *_centroidWeights,
                                            const int *_oneRingScatterAddr,
                                            const float _sigma,
                                            const float _contactAngle)
{

    int tid = threadIdx.x + (blockIdx.x * blockDim.x);

    if(tid >= _numVerts)
    {
        return;
    }


    //----------------------------------------------------
    // initialise variables
    glm::vec3 deformedVert = _deformedVert[tid];
    glm::vec3 deformedNorm = _normal[tid];
    float origIsoValue = _origIsoValue[tid];
    int startNeighAddr = _oneRingScatterAddr[tid];
    int numNeighs = _oneRingScatterAddr[tid+1] - startNeighAddr;
    glm::vec3 newGrad = glm::vec3(0.0f, 0.0f, 0.0f);
    float newIsoValue = 0.0f;


    //----------------------------------------------------
    // Get iso value from global field
    EvalGradGlobalField(newIsoValue, newGrad, deformedVert, _numVerts,
                        _textureSpace, _rigidTransforms, _fieldFuncs, _numFields,
                        _compOps, _theta, _numOps,
                        _compFields, _numCompFields);

    _prevIsoGrad[tid] = newGrad;

    //----------------------------------------------------
    // Perform Tangential Relaxation
    const int *oneRing = (_oneRingVerts + startNeighAddr);
    const float *centroid = (_centroidWeights + startNeighAddr);

    TangentialRelaxation(deformedVert, deformedNorm, origIsoValue, newIsoValue, _deformedVert, oneRing, centroid, numNeighs);


    //----------------------------------------------------
    // Update data
    _deformedVert[tid] = deformedVert;


}


//------------------------------------------------------------------------------------------------

__global__ void GenerateOneRingCentroidWeights_Kernel(glm::vec3 *d_verts,
                                                      const glm::vec3 *d_normals,
                                                      const int _numVerts,
                                                      float *_centroidWeights,
                                                      const int *_oneRingIds,
                                                      const glm::vec3 *_oneRingVerts,
                                                      const int *_numNeighsPerVert,
                                                      const int *_oneRingScatterAddr)
{

    int tid = threadIdx.x + (blockIdx.x * blockDim.x);

    if(tid >= _numVerts)
    {
        return;
    }

    //-------------------------------------------------------------------
    // Passed sanity check lets get down to business

    glm::vec3 v = d_verts[tid];
    glm::vec3 n = d_normals[tid];
    int startNeighAddr = _oneRingScatterAddr[tid];
//    int numNeighs = _numNeighsPerVert[tid];
    int numNeighs = _oneRingScatterAddr[tid+1] - startNeighAddr;


    glm::vec3 oneRingVerts[10];
    glm::vec3 q[10];
    glm::vec3 s[10];
    for(int i=0; i<numNeighs; ++i)
    {
        int neighId = startNeighAddr + i;

        oneRingVerts[i] = _oneRingVerts[neighId];
        q[i] = ProjectPointOnToPlane(oneRingVerts[i], v, n);
        s[i] = q[i] - v;

        _centroidWeights[neighId] = 0.0f;
    }


    float r[10];
    float A[10];
    float D[10];

    // Check for coords close to/on boundary of cage
    for(int i=0; i<numNeighs; ++i)
    {
        int nextI = (i+1)%numNeighs;
        int neighId = startNeighAddr + i;
        int nextNeighId = startNeighAddr + nextI;

        r[i] = glm::length(s[i]);
        glm::vec3 x = glm::cross(s[i], s[nextI]);
        A[i] = 0.5f * glm::length(x);
//        float dot = glm::dot(x, n);
//        A[i] = (dot >= 0.0f) ? A[i] : -A[i];
        D[i] = glm::dot(s[i], s[nextI]);

        if(r[i] < FLT_EPSILON)
        {
            _centroidWeights[neighId] = 1.0f;
            return;
        }
        else if(fabs(A[i]) < FLT_EPSILON && D[i] < 0.0f)
        {
            glm::vec3 dv = q[nextI] - q[i];
            float dl = glm::length(dv);
            // TODO: handle assertions dl==0
            dv = v - q[i];
            float mu = glm::length(dv) / dl;
            // TODO: handle assertions 0<=mu<=1
            _centroidWeights[neighId] = 1.0f - mu;
            _centroidWeights[nextNeighId] = mu;
            return;
        }

    }


    float tanalpha[10]; // tan(alpha/2)
    for( int i = 0; i < numNeighs; ++i)
    {
        int nextI = (i+1)%numNeighs;
        tanalpha[i] = (r[i]*r[nextI] - D[i])/(2.0*A[i]);
    }


    float w[10];
    float W = 0.0f;
    for( int i = 0; i < numNeighs; ++i)
    {
        int prevI = (numNeighs+i-1)%numNeighs; // to avoid potential negative result of % operator

        w[i] = 2.0*( tanalpha[i] + tanalpha[prevI] )/r[i];
        W += w[i];
    }


    if( fabs(W) > 0.0)
    {
        for( int i = 0; i < numNeighs; ++i)
        {
            int neighId = startNeighAddr + i;
            _centroidWeights[neighId] = w[i] / W;
        }
    }
}
