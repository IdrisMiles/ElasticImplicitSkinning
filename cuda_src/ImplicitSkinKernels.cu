#include "hip/hip_runtime.h"
#include "ImplicitSkinKernels.h"
#include <hip/hip_runtime.h>


//------------------------------------------------------------------------------------------------
// CUDA Device Functions
//------------------------------------------------------------------------------------------------

//------------------------------------------------------------------------------------------------

__device__ glm::vec3 ProjectPointOnToPlane(const glm::vec3 &_point, const glm::vec3 &_planeOrigin, const glm::vec3 &_planeNormal)
{
    return (_point - (glm::dot(_point - _planeOrigin, _planeNormal) * _planeNormal));
}

//------------------------------------------------------------------------------------------------

__device__ void VertexProjection(glm::vec3 &_deformedVert,
                                 const float &_origIso,
                                 const float &_newIso,
                                 const glm::vec3 &_newIsoGrad,
                                 glm::vec3 &_prevIsoGrad,
                                 float &_gradAngle,
                                 const float &_sigma,
                                 const float &_contactAngle)
{
    float angle = _gradAngle = glm::degrees(glm::angle(glm::normalize(_newIsoGrad), glm::normalize(_prevIsoGrad)));

    if(angle <= _contactAngle)
    {
        glm::vec3 displacement = ( _sigma * (_newIso - _origIso) * (_newIsoGrad / glm::length2(_newIsoGrad)));
        _deformedVert = _deformedVert + displacement;
    }
}

//------------------------------------------------------------------------------------------------

__device__ void TangentialRelaxation (glm::vec3 &_deformedVert,
                                     const glm::vec3 &_normal,
                                     const float _origIso,
                                     const float _newIso,
                                      glm::vec3 *_verts,
                                     const int *_oneRingNeigh,
                                     const float *_centroidWeights,
                                     const int _numNeighs)
{
    float mu = 1.0f - powf(fabs(_newIso- _origIso) - 1.0f, 4.0f);
    mu = max(mu, 0.0f);

    glm::vec3 norm(0.0f, 0.0f, 0.0f);
    for(int i=0; i<_numNeighs; i++)
    {
        int nextNeigh = ((i+1)%_numNeighs);
        glm::vec3 neighVert = _verts[_oneRingNeigh[i]];
        glm::vec3 nextNeighVert = _verts[_oneRingNeigh[nextNeigh]];
        glm::vec3 faceNorm = glm::cross(neighVert - _deformedVert, nextNeighVert-_deformedVert);
        norm += faceNorm;
    }
    norm = glm::normalize(norm);

    glm::vec3 sumWeightedCentroid(0.0f);
    for(int i=0; i<_numNeighs; i++)
    {
        glm::vec3 neighVert = _verts[_oneRingNeigh[i]];
        glm::vec3 projNeighVert = ProjectPointOnToPlane(neighVert, _deformedVert, norm);//_normal);
        float barycentricCoord = _centroidWeights[i];
        sumWeightedCentroid += barycentricCoord * projNeighVert;
    }

    _deformedVert = ((1.0f - mu) * _deformedVert) + (mu * sumWeightedCentroid);
}

//------------------------------------------------------------------------------------------------

__device__ void LaplacianSmoothing(glm::vec3 &_deformedVert,
                                   const glm::vec3 &_normal,
                                   const int *_oneRingNeigh,
                                   const float *_centroidWeights,
                                   const int _numNeighs,
                                   const glm::vec3 *_verts,
                                   const float _beta)
{
    glm::vec3 centroid(0.0f, 0.0f, 0.0f);

    for(int i=0; i<_numNeighs; i++)
    {
        centroid += (_centroidWeights[i] * _verts[_oneRingNeigh[i]]);
    }

    _deformedVert = ((1.0f - _beta) * _deformedVert) + (_beta * centroid);
}

//------------------------------------------------------------------------------------------------

__device__ void EvalGlobalField(float &_outputF,
                                const glm::vec3 &_samplePoint,
                                const uint _numSamples,
                                const glm::mat4 *_textureSpace,
                                const glm::mat4 *_rigidTransforms,
                                const hipTextureObject_t *_fieldFuncs,
                                const uint _numFields,
                                const hipTextureObject_t *_compOps,
                                const hipTextureObject_t *_theta,
                                const uint _numOps,
                                const ComposedFieldCuda *_compFields,
                                const uint _numCompFields)
{
    float f[100];
    glm::vec3 df[100];
    int i=0;
    for(i=0; i<_numFields; i++)
    {
        glm::mat4 rigidTrans = _rigidTransforms[i];
        glm::mat4 textureSpace = _textureSpace[i];
        glm::vec3 transformedPoint = glm::vec3(glm::inverse(rigidTrans) * glm::vec4(_samplePoint, 1.0f));
        glm::vec3 texturePoint = glm::vec3(textureSpace * glm::vec4(transformedPoint, 1.0f));
        texturePoint = 1.015f*texturePoint;

        float4 val = tex3D<float4>(_fieldFuncs[i], texturePoint.x, texturePoint.y, texturePoint.z);
        f[i] = val.w;
        df[i] = glm::vec3(val.x, val.y, val.z);
    }


    float cf[100];
    float maxF = FLT_MIN;
    for(i=0; i<_numCompFields; i++)
    {
        int f1Id = _compFields[i].fieldFuncA;
        int f2Id = _compFields[i].fieldFuncB;
        int coId = _compFields[i].compOp;

        glm::vec3 df1(df[f1Id].x, df[f1Id].y, df[f1Id].z);
        glm::vec3 df2(df[f2Id].x, df[f2Id].y, df[f2Id].z);
        float angle = glm::angle(df1, df2);
        float theta = tex1D<float>(_theta[coId], (angle*0.5f*M_1_PI));

        float4 val = tex3D<float4>(_compOps[coId], f[f1Id], f[f2Id], theta);
        cf[i] = (f2Id < 0) ? f[f1Id] : val.w;

        maxF = (cf[i]>maxF) ? cf[i] : maxF;
    }


    _outputF = maxF;

}

//------------------------------------------------------------------------------------------------


__device__ void EvalGradGlobalField(float &_outputF,
                                glm::vec3 &_outputG,
                                const glm::vec3 &_samplePoint,
                                const uint _numSamples,
                                const glm::mat4 *_textureSpace,
                                const glm::mat4 *_rigidTransforms,
                                const hipTextureObject_t *_fieldFuncs,
                                const uint _numFields,
                                const hipTextureObject_t *_compOps,
                                const hipTextureObject_t *_theta,
                                const uint _numOps,
                                const ComposedFieldCuda *_compFields,
                                const uint _numCompFields)
{
    float f[100];
    glm::vec3 df[100];
    int i=0;
    for(i=0; i<_numFields; i++)
    {
        glm::mat4 rigidTrans = _rigidTransforms[i];
        glm::mat4 textureSpace = _textureSpace[i];
        glm::vec3 transformedPoint = glm::vec3(glm::inverse(rigidTrans) * glm::vec4(_samplePoint, 1.0f));
        glm::vec3 texturePoint = glm::vec3(textureSpace * glm::vec4(transformedPoint, 1.0f));
        texturePoint = 1.015f*texturePoint;

        float4 val = tex3D<float4>(_fieldFuncs[i], texturePoint.x, texturePoint.y, texturePoint.z);
        f[i] = val.w;
        df[i] = glm::vec3(val.x, val.y, val.z);
    }


    float cf[100];
    glm::vec3 cdf[100];
    glm::vec3 grad;
    float maxF = FLT_MIN;
    for(i=0; i<_numCompFields; i++)
    {
        int f1Id = _compFields[i].fieldFuncA;
        int f2Id = _compFields[i].fieldFuncB;
        int coId = _compFields[i].compOp;

        glm::vec3 f1Grad(df[f1Id].x, df[f1Id].y, df[f1Id].z);
        glm::vec3 f2Grad(df[f2Id].x, df[f2Id].y, df[f2Id].z);
        float angle = glm::angle(f1Grad, f2Grad);
        float theta = tex1D<float>(_theta[coId], (angle*0.5f*M_1_PI));

        // composed field value
        float4 val = tex3D<float4>(_compOps[coId], f[f1Id], f[f2Id], theta);
        cf[i] = (f2Id < 0) ? f[f1Id] : val.w;

        // compose field gradient
        float df1 = tex3D<float4>(_compOps[coId], f[f1Id]+0.1f, f[f2Id], theta).w - cf[i];
        float df2 = tex3D<float4>(_compOps[coId], f[f1Id], f[f2Id]+0.1f, theta).w - cf[i];
        cdf[i] = (df[f1Id]*df1) + (df[f2Id]*df2);

        // apply max operator
        grad = (cf[i]>maxF) ? cdf[i] : grad;
        maxF = (cf[i]>maxF) ? cf[i] : maxF;
    }


    _outputF = maxF;
    _outputG = grad;

}



//------------------------------------------------------------------------------------------------
// CUDA Global Kernels
//------------------------------------------------------------------------------------------------

__global__ void EvalGlobalField_Kernel(float *_output,
                                       const glm::vec3 *_samplePoint,
                                       const uint _numSamples,
                                       const glm::mat4 *_textureSpace,
                                       const glm::mat4 *_rigidTransforms,
                                       const hipTextureObject_t *_fieldFuncs,
                                       const uint _numFields,
                                       const hipTextureObject_t *_compOps,
                                       const hipTextureObject_t *_theta,
                                       const uint _numOps,
                                       const ComposedFieldCuda *_compFields,
                                       const uint _numCompFields)
{
    int tid = threadIdx.x + (blockIdx.x * blockDim.x);

    if(tid >= _numSamples)
    {
        return;
    }

    EvalGlobalField(_output[tid], _samplePoint[tid], _numSamples,
                    _textureSpace, _rigidTransforms, _fieldFuncs, _numFields,
                    _compOps, _theta, _numOps,
                    _compFields, _numCompFields);
}

//------------------------------------------------------------------------------------------------

__global__ void EvalGradGlobalField_Kernel(float *_output,
                                           glm::vec3 *_outputG,
                                       const glm::vec3 *_samplePoint,
                                       const uint _numSamples,
                                       const glm::mat4 *_textureSpace,
                                       const glm::mat4 *_rigidTransforms,
                                       const hipTextureObject_t *_fieldFuncs,
                                       const uint _numFields,
                                       const hipTextureObject_t *_compOps,
                                       const hipTextureObject_t *_theta,
                                       const uint _numOps,
                                       const ComposedFieldCuda *_compFields,
                                       const uint _numCompFields)
{
    int tid = threadIdx.x + (blockIdx.x * blockDim.x);

    if(tid >= _numSamples)
    {
        return;
    }

    EvalGradGlobalField(_output[tid], _outputG[tid], _samplePoint[tid], _numSamples,
                        _textureSpace, _rigidTransforms, _fieldFuncs, _numFields,
                        _compOps, _theta, _numOps,
                        _compFields, _numCompFields);
}


//------------------------------------------------------------------------------------------------

__global__ void LinearBlendWeightSkin_Kernel(glm::vec3 *_deformedVert,
                                             const glm::vec3 *_origVert,
                                             glm::vec3 *_deformedNorms,
                                             const glm::vec3 *_origNorms,
                                             const glm::mat4 *_transform,
                                             const uint *_boneId,
                                             const float *_weight,
                                             const uint _numVerts,
                                             const uint _numBones)
{
    int tid = threadIdx.x + (blockIdx.x * blockDim.x);

    if(tid >= _numVerts)
    {
        return;
    }

    glm::mat4 boneTransform = glm::mat4(0.0f);

    float totalWeight = 0.0f;
    for(int i=0; i<4; i++)
    {
        unsigned int boneId = _boneId[(tid*4) + i];
        float w = _weight[(tid*4) + i];
        boneTransform += (_transform[boneId] * w);

        totalWeight+=w;
    }

    _deformedVert[tid] = glm::vec3(boneTransform * glm::vec4(_origVert[tid], 1.0f));
    _deformedNorms[tid] = glm::transpose(glm::inverse(glm::mat3(boneTransform))) * _origNorms[tid];
}

//------------------------------------------------------------------------------------------------

__global__ void VertexProjection_Kernel(glm::vec3 *_deformedVert,
                                        const glm::vec3 *_normal,
                                        const float *_origIsoValue,
                                        glm::vec3 *_prevIsoGrad,
                                        const uint _numVerts,
                                        const glm::mat4 *_textureSpace,
                                        const glm::mat4 *_rigidTransforms,
                                        const hipTextureObject_t *_fieldFuncs,
                                        const uint _numFields,
                                        const hipTextureObject_t *_compOps,
                                        const hipTextureObject_t *_theta,
                                        const uint _numOps,
                                        const ComposedFieldCuda *_compFields,
                                        const uint _numCompFields,
                                        const float _sigma,
                                        const float _contactAngle)
{

    int tid = threadIdx.x + (blockIdx.x * blockDim.x);

    if(tid >= _numVerts)
    {
        return;
    }

    //----------------------------------------------------
    // Get iso value from global field
    glm::vec3 deformedVert = _deformedVert[tid];
    float origIsoValue = _origIsoValue[tid];
    glm::vec3 prevGrad = _prevIsoGrad[tid];
    glm::vec3 newGrad;
    float newIsoValue;

    EvalGradGlobalField(newIsoValue, newGrad, deformedVert, _numVerts,
                        _textureSpace, _rigidTransforms, _fieldFuncs, _numFields,
                        _compOps, _theta, _numOps,
                        _compFields, _numCompFields);


    //----------------------------------------------------
    // Perform vertex projection along gradient of global field
    float gradAngle;

    VertexProjection(deformedVert, origIsoValue, newIsoValue, newGrad, prevGrad, gradAngle, _sigma, _contactAngle);
    prevGrad = newGrad;

    _deformedVert[tid] = deformedVert;
    _prevIsoGrad[tid] = newGrad;

}

//------------------------------------------------------------------------------------------------

__global__ void TangentialRelaxation_Kernel(glm::vec3 *_deformedVert,
                                            const glm::vec3 *_normal,
                                            const float *_origIsoValue,
                                            glm::vec3 *_prevIsoGrad,
                                            const uint _numVerts,
                                            const glm::mat4 *_textureSpace,
                                            const glm::mat4 *_rigidTransforms,
                                            const hipTextureObject_t *_fieldFuncs,
                                            const uint _numFields,
                                            const hipTextureObject_t *_compOps,
                                            const hipTextureObject_t *_theta,
                                            const uint _numOps,
                                            const ComposedFieldCuda *_compFields,
                                            const uint _numCompFields,
                                            const int *_oneRingVerts,
                                            const float *_centroidWeights,
                                            const int *_oneRingScatterAddr,
                                            const float _sigma,
                                            const float _contactAngle)
{

    int tid = threadIdx.x + (blockIdx.x * blockDim.x);

    if(tid >= _numVerts)
    {
        return;
    }


    //----------------------------------------------------
    // initialise variables
    glm::vec3 deformedVert = _deformedVert[tid];
    glm::vec3 deformedNorm = _normal[tid];
    float origIsoValue = _origIsoValue[tid];
    int startNeighAddr = _oneRingScatterAddr[tid];
    int numNeighs = _oneRingScatterAddr[tid+1] - startNeighAddr;
    glm::vec3 newGrad;
    float newIsoValue;


    //----------------------------------------------------
    // Get iso value from global field
    EvalGradGlobalField(newIsoValue, newGrad, deformedVert, _numVerts,
                        _textureSpace, _rigidTransforms, _fieldFuncs, _numFields,
                        _compOps, _theta, _numOps,
                        _compFields, _numCompFields);

    _prevIsoGrad[tid] = newGrad;

    //----------------------------------------------------
    // Perform Tangential Relaxation
    const int *oneRing = (_oneRingVerts + startNeighAddr);
    const float *centroid = (_centroidWeights + startNeighAddr);

    TangentialRelaxation(deformedVert, deformedNorm, origIsoValue, newIsoValue, _deformedVert, oneRing, centroid, numNeighs);


    //----------------------------------------------------
    // Update data
    _deformedVert[tid] = deformedVert;


}


//------------------------------------------------------------------------------------------------

__global__ void GenerateOneRingCentroidWeights_Kernel(glm::vec3 *d_verts,
                                                      const glm::vec3 *d_normals,
                                                      const uint _numVerts,
                                                      float *_centroidWeights,
                                                      const int *_oneRingIds,
                                                      const glm::vec3 *_oneRingVerts,
                                                      const int *_numNeighsPerVert,
                                                      const int *_oneRingScatterAddr)
{

    int tid = threadIdx.x + (blockIdx.x * blockDim.x);

    if(tid >= _numVerts)
    {
        return;
    }

    //-------------------------------------------------------------------
    // Passed sanity check lets get down to business

    glm::vec3 v = d_verts[tid];
    glm::vec3 n = d_normals[tid];
    int startNeighAddr = _oneRingScatterAddr[tid];
//    int numNeighs = _numNeighsPerVert[tid];
    int numNeighs = _oneRingScatterAddr[tid+1] - startNeighAddr;


    glm::vec3 oneRingVerts[10];
    glm::vec3 q[10];
    glm::vec3 s[10];
    for(int i=0; i<numNeighs; ++i)
    {
        int neighId = startNeighAddr + i;

        oneRingVerts[i] = _oneRingVerts[neighId];
        q[i] = ProjectPointOnToPlane(oneRingVerts[i], v, n);
        s[i] = q[i] - v;

        _centroidWeights[neighId] = 0.0f;
    }


    float r[10];
    float A[10];
    float D[10];

    // Check for coords close to/on boundary of cage
    for(int i=0; i<numNeighs; ++i)
    {
        int nextI = (i+1)%numNeighs;
        int neighId = startNeighAddr + i;
        int nextNeighId = startNeighAddr + nextI;

        r[i] = glm::length(s[i]);
        glm::vec3 x = glm::cross(s[i], s[nextI]);
        A[i] = 0.5f * glm::length(x);
//        float dot = glm::dot(x, n);
//        A[i] = (dot >= 0.0f) ? A[i] : -A[i];
        D[i] = glm::dot(s[i], s[nextI]);

        if(r[i] < FLT_EPSILON)
        {
            _centroidWeights[neighId] = 1.0f;
            return;
        }
        else if(fabs(A[i]) < FLT_EPSILON && D[i] < 0.0f)
        {
            glm::vec3 dv = q[nextI] - q[i];
            float dl = glm::length(dv);
            // TODO: handle assertions dl==0
            dv = v - q[i];
            float mu = glm::length(dv) / dl;
            // TODO: handle assertions 0<=mu<=1
            _centroidWeights[neighId] = 1.0f - mu;
            _centroidWeights[nextNeighId] = mu;
            return;
        }

    }


    float tanalpha[10]; // tan(alpha/2)
    for( int i = 0; i < numNeighs; ++i)
    {
        int nextI = (i+1)%numNeighs;
        tanalpha[i] = (r[i]*r[nextI] - D[i])/(2.0*A[i]);
    }


    float w[10];
    float W = 0.0f;
    for( int i = 0; i < numNeighs; ++i)
    {
        int prevI = (numNeighs+i-1)%numNeighs; // to avoid potential negative result of % operator

        w[i] = 2.0*( tanalpha[i] + tanalpha[prevI] )/r[i];
        W += w[i];
    }


    if( fabs(W) > 0.0)
    {
        for( int i = 0; i < numNeighs; ++i)
        {
            int neighId = startNeighAddr + i;
            _centroidWeights[neighId] = w[i] / W;
        }
    }
}
