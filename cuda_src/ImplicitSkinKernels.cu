#include "hip/hip_runtime.h"
#include "ImplicitSkinKernels.cuh"


__global__ void EvaluateGlobalField(float *_output,
                                    glm::vec3 *_samplePoint,
                                    uint _numSamples,
                                    glm::mat4 _textureSpace,
                                    glm::mat4 *_rigidTransforms,
                                    hipTextureObject_t *_fieldFuncs,
                                    hipTextureObject_t *_fieldDeriv,
                                    uint _numFields,
                                    hipTextureObject_t *_compOps,
                                    hipTextureObject_t *_theta, // opening function
                                    uint _numOps,
                                    CompField *_compFields,
                                    uint _numCompFields)
{
    int tid = threadIdx.x + (blockIdx.x * blockDim.x);

    if(tid >= _numSamples)
    {
        return;
    }


    glm::vec3 samplePoint = _samplePoint[tid];
    glm::mat4 textureSpace = _textureSpace;


    float maxF = FLT_MIN;
    float f[100];
//    float3 df[100];
    int i=0;
    for(i=0; i<_numFields; i++)
    {
        glm::vec3 transformedPoint = _rigidTransforms[i] * glm::vec4(samplePoint,1.0f);
        glm::vec3 texturePoint = textureSpace * glm::vec4(transformedPoint, 1.0f);

        f[i] = tex3D<float>(_fieldFuncs[i], texturePoint.x, texturePoint.y, texturePoint.z);
//        df[i] = tex3D<float3>(_fieldDeriv[i], texturePoint.x, texturePoint.y, texturePoint.z);

        maxF = (f[i]>maxF) ? f[i] : maxF;
    }


//    float cf[100];
//    float maxF = FLT_MIN;
//    for(i=0; i<_numCompFields; i++)
//    {
//        int f1Id = _compFields[i].fieldFuncA;
//        int f2Id = _compFields[i].fieldFuncB;
//        int coId = _compFields[i].compOp;

//        glm::vec3 df1(df[f1Id].x, df[f1Id].y, df[f1Id].z);
//        glm::vec3 df2(df[f2Id].x, df[f2Id].y, df[f2Id].z);
//        float angle = glm::angle(df1, df2);
//        float theta = tex1D<float>(_theta, angle*0.5f*M_1_PI);

//        cf[i] = tex3D<float>(_compOps[coId], f[f1Id], f[f2Id], theta);

//        maxF = (cf[i]>maxF) ? cf[i] : maxF;
//    }


    _output[tid] = maxF;
}

//------------------------------------------------------------------------------------------------

__global__ void LinearBlendWeightSkin(glm::vec3 *_deformedVert,
                                      const glm::vec3 *_origVert,
                                      const glm::mat4 *_transform,
                                      const uint *_boneId,
                                      const float *_weight,
                                      const uint _numVerts,
                                      const uint _numBones)
{
    int tid = threadIdx.x + (blockIdx.x * blockDim.x);

    if(tid >= _numVerts)
    {
        return;
    }

    glm::mat4 boneTransform = glm::mat4(0.0f);

    float totalWeight = 0.0f;
    for(int i=0; i<4; i++)
    {
        unsigned int boneId = _boneId[(tid*4) + i];
        float w = _weight[(tid*4) + i];
        boneTransform += (_transform[boneId] * w);

        totalWeight+=w;
    }

    _deformedVert[tid] = boneTransform * glm::vec4(_origVert[tid], 1.0f);

}

