#include "implicitskindeformer.h"
#include "ImplicitSkinKernels.h"
#include "hip/hip_runtime_api.h"



//------------------------------------------------------------------------

ImplicitSkinDeformer::ImplicitSkinDeformer(const Mesh _origMesh,
                                         const GLuint _meshVBO,
                                         const std::vector<glm::mat4> &_transform):
    m_init(false),
    m_meshDeformedMapped(false)
{
    m_numVerts = _origMesh.m_meshVerts.size();

    // Get bone ID and weights per vertex
    unsigned int boneIds[m_numVerts *4];
    float weights[m_numVerts *4];
    int i=0;
    for(auto &bw : _origMesh.m_meshBoneWeights)
    {
        float totalW = 0.0f;
        for(int j=0; j<4; j++)
        {
            boneIds[i+j] = bw.boneID[j];
            weights[i+j] = bw.boneWeight[j];
            totalW += bw.boneWeight[j];
        }

        // Normalize weights
        if(totalW < 1.0f)
        {
            for(int j=0; j<4; j++)
            {
                weights[i+j] /= totalW;
            }
        }
        i+=4;
    }


    if(!m_init)
    {
        checkCudaErrors(hipSetDevice(0));

        // Register vertex buffer with CUDA
        checkCudaErrors(hipGraphicsGLRegisterBuffer(&m_meshVBO_CUDA, _meshVBO, cudaGraphicsMapFlagsWriteDiscard));

        // Allocate cuda memory
        checkCudaErrorsMsg(hipMalloc(&d_meshOrigPtr, m_numVerts * sizeof(glm::vec3)), "Allocate memory for original mesh");
        checkCudaErrorsMsg(hipMalloc(&d_transformPtr, _transform.size() * sizeof(glm::mat4)), "Allocate memory for transforms");
        checkCudaErrorsMsg(hipMalloc(&d_boneIdPtr, m_numVerts * 4 * sizeof(unsigned int)), "Allocate memory for bone Ids");
        checkCudaErrorsMsg(hipMalloc(&d_weightPtr, m_numVerts * 4 * sizeof(float)), "Allocate memory for bone weights");

        // copy memory over to cuda
        checkCudaErrors(hipMemcpy((void*)d_meshOrigPtr, (void*)&_origMesh.m_meshVerts[0], m_numVerts * sizeof(glm::vec3), hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy((void*)d_transformPtr, (void*)&_transform[0][0][0], _transform.size() * sizeof(glm::mat4), hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy((void*)d_boneIdPtr, (void*)boneIds, m_numVerts *4* sizeof(unsigned int), hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy((void*)d_weightPtr, (void*)weights, m_numVerts *4* sizeof(float), hipMemcpyHostToDevice));

        checkCudaErrors(hipDeviceSynchronize());
    }
    m_init = true;
}

//------------------------------------------------------------------------------------------------

ImplicitSkinDeformer::~ImplicitSkinDeformer()
{

    if(m_init)
    {
        checkCudaErrors(hipGraphicsUnregisterResource(m_meshVBO_CUDA));
        checkCudaErrors(hipFree(d_meshOrigPtr));
        checkCudaErrors(hipFree(d_transformPtr));
        checkCudaErrors(hipFree(d_boneIdPtr));
        checkCudaErrors(hipFree(d_weightPtr));
        m_init = false;
    }
}

//------------------------------------------------------------------------------------------------

void ImplicitSkinDeformer::PerformLBWSkinning(const std::vector<glm::mat4> &_transform)
{

    if(!m_init)
    {
        return;
    }

    checkCudaErrors(hipMemcpy((void*)d_transformPtr, &_transform[0][0][0], _transform.size() * sizeof(glm::mat4), hipMemcpyHostToDevice));

    LinearBlendWeightSkin(GetMeshDeformedPtr(),
                           d_meshOrigPtr,
                           d_transformPtr,
                           d_boneIdPtr,
                           d_weightPtr,
                           m_numVerts,
                           _transform.size());

    getLastCudaError("LinearBlendWeightSkin Failed");

    checkCudaErrors(hipDeviceSynchronize());
    ReleaseMeshDeformedPtr();
}

//------------------------------------------------------------------------------------------------

glm::vec3 *ImplicitSkinDeformer::GetMeshDeformedPtr()
{
    if(!m_meshDeformedMapped)
    {
        size_t numBytes;
        checkCudaErrors(hipGraphicsMapResources(1, &m_meshVBO_CUDA, 0));
        checkCudaErrors(hipGraphicsResourceGetMappedPointer((void **)&d_meshDeformedPtr, &numBytes, m_meshVBO_CUDA));

        m_meshDeformedMapped = true;
    }

    return d_meshDeformedPtr;
}

//------------------------------------------------------------------------------------------------

void ImplicitSkinDeformer::ReleaseMeshDeformedPtr()
{
    if(m_meshDeformedMapped)
    {
        checkCudaErrors(hipGraphicsUnmapResources(1, &m_meshVBO_CUDA, 0));
        m_meshDeformedMapped = false;
    }
}
